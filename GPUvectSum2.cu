#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

 #define N 5

 __global__ void Add(int *a, int*b, int *c){
 int i = blockIdx.x; 
 if(i<N){
 c[i] = a[i] + b[i];   
 }
}

 int main(){
 int a[N] = {1,2,3,4,5}, b[N] = {5,6,7,8,9}; 
 int c[N];
 int *dev_a, *dev_b, *dev_c; 


 hipMalloc((void**)&dev_a, N*sizeof(int));
 hipMalloc((void**)&dev_b, N*sizeof(int));
 hipMalloc((void**)&dev_c, N*sizeof(int));


 hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice); 
 hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

 Add<<<2,1>>>(dev_a, dev_b, dev_c); // HERE IS THE CRITICAL LINE !!!!!!


 hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);


 int i; printf("c[i] = ");
 for(i=0;i<N;i++){
    printf("%d ", c[i]);
 }


 hipFree(dev_a); 
 hipFree(dev_b); 
 hipFree(dev_c);

 printf("\n");
 return 0;
 }