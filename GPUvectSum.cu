#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <gsl/gsl_cblas.h>
#include <hip/hip_runtime.h>

#define N   10


__global__ void Add(int *a, int*b, int *c){
 int i = blockIdx.x; 
 if(i<N){
 c[i] = a[i] + b[i];   
 }
}





int main(void) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	
	//allocate the memory on the GPU
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	
	//fill the array's 'a' and 'b' on the CPU
	for (int i=0; i<N; i++){
		a[i] = -i;
		b[i] = i * i;
	}
	
	//copy the arrays 'a' and 'b' to the GPU
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);
	
	double rMean[8] = {3.15,1.75,-6.4,-2.9,-6.8,-0.54,-6.8,-5.3};
	double covMatrix[8][8] = {
	  {0.001005, 0.001328, -0.000579, -0.000675, 0.000121, 0.000128, -0.000445, -0.000437},
	  {0.001328, 0.007277, -0.001307, -0.00061, -0.002237, -0.000989, 0.001442, -0.001535},	
	  {-0.000579, -0.001307, 0.059852, 0.027588, 0.063497, 0.023036, 0.032967, 0.048039},
	  {-0.000675, -0.00061, 0.027588, 0.029609, 0.026572, 0.021465, 0.020697, 0.029854}, 
	  {0.000121, -0.002237, 0.063497, 0.026572, 0.102488, 0.042744, 0.039943, 0.065994}, 
	  {0.000128, -0.000989, 0.023036, 0.021465, 0.042744, 0.032056, 0.019881, 0.032235}, 
	  {-0.000445, 0.001442, 0.032967, 0.020697, 0.039943, 0.019881, 0.028355, 0.035064},
	  {-0.000437, -0.001535, 0.048039, 0.029854, 0.065994, 0.032235, 0.035064, 0.079958}
	};
	
	
	
	Add<<<N,1>>>( dev_a, dev_b, dev_c);
	
	
	//copy the array 'c' back from the GPU to the CPU
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	
	// display the results
	int i; 
	for(i=0; i<N; i++){
		printf("%d", a[i], "%d",  b[i], "%d",  c[i], "\r\n");
	}
	
	// free the memory allocated on the GPU
	hipFree( dev_a);
	hipFree( dev_b);
	hipFree( dev_c);
	
	return 0;
}